#include "hip/hip_runtime.h"
#include "product/product.h"
#include "matrix/formats/matrixEllpack.h"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

#define BD 32

const dim3 BLOCK_DIM(BD,BD);

/**
 * Per come è implementato attualmente il kernel ogni thread si prende una riga della prima matrice e la moltuplica per tutte le colonne
 * della seconda matrice. Si parte da questa versione base e si introducono le varie ottimizzazioni.
*/
__global__ void gpuMatrixMultiVectorELL(int rowsA, int colsA, int colsMulti, const double* A_values ,const int* A_cols, const double* multiVect, double* y) {
    
    //Indici del blocco.
    int bx = blockIdx.x;
    int by = blockIdx.y;

    //Indici del thread
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Individuo inizio e fine della sottomatrice da utilizzare.
    int aBegin = colsA * BD * by;
    int aEnd   = aBegin + colsA - 1;
    int aStep  = BD;

    // Individuo inizio e fine della sottomatrice da utilizzare
    int bBegin = BD* bx;
    int bStep  = BD* colsMulti;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    for (int a = aBegin, b = bBegin;a <= aEnd;a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BD][BD];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BD][BD];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A_values[a + colsA * ty + tx];
    Bs[ty][tx] = multiVect[b + colsMulti * ty + A_cols[a + colsA * ty + tx]];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix

    for (int k = 0; k < BD; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();

    int c = colsMulti * BD * by + BD * bx;
    y[c + colsMulti * ty + tx] = Csub;
  }


}

/**
 * Converte una matrice di double in un array di double
 */
void convert2Dto1DDouble(double** mat, double* vet,int rows,int cols){
     for(int i=0,k=0; i<rows; i++)
        for(int j=0; j<cols;j++, k++)
            vet[k] = mat[i][j];
}

/**
 * Converte una matrice di inter in un array di inter
 */
void convert2Dto1DInt(int** mat, int* vet,int rows,int cols){
     for(int i=0,k=0; i<rows; i++)
        for(int j=0; j<cols;j++, k++)
            vet[k] = mat[i][j];
}

/**
 * Converte un vettore in una matrice in formato Matrix. 
 * TODO: potrebbe avere senso implementare questa funzione direttamente in Matrix
 */
void convertToMatrixFormat(double* h_y, Matrix *mResult){
    for(int i=0; i<mResult->rows ; i++)
        for(int j=0; j<mResult->cols ; j++){
            mResult->put(mResult,i,j,h_y[i*mResult->cols+ j]);
        }    
}

/**
 * Funzione definita in product.h che invoca il kernel sulla GPU e raccoglie dati sulle prestazioni.
 */
int productMatrixMatrixParallelEllpack(Matrix *matrix1, Matrix *matrix2, Matrix *mResult, Sample *sample){
    
    DataEllpack * dataEllpack = (DataEllpack*) matrix1->data;
    double** multiVector = (double**) matrix2->data;

    //Strutture usate nella misurazione delle prestazioni
    struct timespec  tStart;
    struct timespec  tEnd;

    int dimMatrix = dataEllpack->colsSubMat * dataEllpack->rowsSubMat;
    int dimMulti  = matrix2->cols * matrix2->rows;
    int dimResult = dataEllpack->rowsSubMat * matrix2->cols;

    // ---------------------- Host memory initialisation ---------------------- //

    double  *h_A_values   = (double *) calloc(dimMatrix, sizeof(double));
    double  *h_Multi_Vec  = (double *) calloc(dimMulti , sizeof(double));
    double  *h_y          = (double *) calloc(dimResult, sizeof(double));
    int     *h_A_cols     = (int *)    calloc(dimMatrix, sizeof(int));



    convert2Dto1DDouble(dataEllpack->matValues,h_A_values  ,dataEllpack->rowsSubMat, dataEllpack->colsSubMat);
    convert2Dto1DDouble(multiVector           ,h_Multi_Vec ,matrix2->rows, matrix2->cols);
    convert2Dto1DInt   (dataEllpack->matCols  ,h_A_cols    ,dataEllpack->rowsSubMat, dataEllpack->colsSubMat);


    // ---------------------- Device memory initialisation ---------------------- //

    double  *d_A_values;
    double  *d_Multi_Vec;
    double  *d_y;
    int     *d_A_cols;

    checkCudaErrors(hipMalloc((void**) &d_A_values , dimMatrix*sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_Multi_Vec, dimMulti *sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_y        , dimResult*sizeof(double)));
    checkCudaErrors(hipMalloc((void**) &d_A_cols   , dimMatrix*sizeof(int)));

    checkCudaErrors(hipMemcpy(d_A_values   , h_A_values,   dimMatrix * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Multi_Vec  , h_Multi_Vec,  dimMulti  * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y          , h_y,          dimResult * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_A_cols     , h_A_cols,     dimMatrix * sizeof(int)   , hipMemcpyHostToDevice));


    // ---------------------- GPU ---------------------- //

    dim3 GRID_DIM(matrix2->cols / BLOCK_DIM.x, dataEllpack->rowsSubMat / BLOCK_DIM.y);

    clock_gettime(CLOCK_REALTIME,&tStart);
    gpuMatrixMultiVectorELL<<<GRID_DIM, BLOCK_DIM >>>(dataEllpack->rowsSubMat, dataEllpack->colsSubMat,matrix2->cols ,d_A_values, d_A_cols, d_Multi_Vec,d_y);
    clock_gettime(CLOCK_REALTIME ,&tEnd);      

    checkCudaErrors(hipDeviceSynchronize());


    // ---------------------- Collect result ---------------------- //

    hipMemcpy(h_y, d_y, dimResult* sizeof(double), hipMemcpyDeviceToHost);
    convertToMatrixFormat(h_y, mResult); 

    sample->execTimeSecs = tEnd.tv_sec - tStart.tv_sec;
    sample->execTimeNsecs = tEnd.tv_nsec - tStart.tv_nsec;
    sample->productName = (char *)__func__;
    return 0;
}


